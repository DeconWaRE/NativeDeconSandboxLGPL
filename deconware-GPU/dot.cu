
#include <stdio.h>
#include <hip/hip_runtime.h>

#define imin(a,b) (a<b?a:b)

const int threadsPerBlock = 256;

__global__ void dot( float *a, float *b, float *c, int N ) {
    __shared__ float cache[threadsPerBlock];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;

    float   temp = 0;
    while (tid < N) {
        temp += a[tid] * b[tid];
        tid += blockDim.x * gridDim.x;
    }
    
    // set the cache values
    cache[cacheIndex] = temp;
    
    // synchronize threads in this block
    __syncthreads();

    // for reductions, threadsPerBlock must be a power of 2
    // because of the following code
    int i = blockDim.x/2;
    while (i != 0) {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}

extern "C" {
float dot_device(size_t N, float *in1, float *in2);
}

float dot_device(size_t N, float *in1, float *in2) 
{
	fprintf(stderr, "Dot device %d\n", N);
	float out=0.0;

	float *partial_out;
	float   *dev_in1, *dev_in2, *dev_partial_out;

	hipError_t err;

	fprintf(stderr,"N %d\n", N);

	hipDeviceReset();

	// calculate blocks per grid
	const int blocksPerGrid = imin( N, (128+threadsPerBlock-1) / threadsPerBlock );

	// allocate memory

	fprintf(stderr, "allocate memory\n");

	err = hipMalloc(&dev_in1, N*sizeof(float));
	if(err) goto cudaErr;
    
	err = hipMalloc(&dev_in2, N*sizeof(float));
	if(err) goto cudaErr;

	// allocate memory for the partial result on the GPU
	err = hipMalloc( (void**)&dev_partial_out, blocksPerGrid*sizeof(float) );
	if(err) goto cudaErr;

	// allocate memory for the partial result on the CPU
	partial_out = (float*)malloc( blocksPerGrid*sizeof(float) );

	fprintf(stderr, "transfer memory\n");

	// transfer memory from cpu to gpu
 	err = hipMemcpy(dev_in1, in1, N*sizeof(float), hipMemcpyHostToDevice);
    if(err) goto cudaErr;
    
    err = hipMemcpy(dev_in2, in2, N*sizeof(float), hipMemcpyHostToDevice);
    if(err) goto cudaErr;

	fprintf(stderr, "run kernel\n");

	// run the kernel
	dot<<<blocksPerGrid,threadsPerBlock>>>( dev_in1, dev_in2, dev_partial_out, N);

	// copy the partial reduction back from the GPU
 	hipMemcpy( partial_out, dev_partial_out,
                              blocksPerGrid*sizeof(float),
                              hipMemcpyDeviceToHost );

 	// finish up on the CPU side
    for (int i=0; i<blocksPerGrid; i++) {
        out += partial_out[i];
    }

	cudaErr:
	    fprintf(stderr, "Dot Device: CUDA error: %d\n", err);
	    goto cleanup;

	cleanup:
		if(dev_in1) hipFree(dev_in1);
	    if(dev_in2) hipFree(dev_in2);
		if(dev_partial_out) hipFree(dev_partial_out);
		free( partial_out );
	    hipDeviceReset();
	    return out;
}



